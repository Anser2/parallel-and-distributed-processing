// filepath: OpenCL_CUDA/laplace_solver/laplace_cuda.cu
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

__global__ void laplace_step(float* grid, float* new_grid, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i > 0 && i < N-1 && j > 0 && j < N-1) {
        new_grid[i*N + j] = 0.25f * (grid[(i-1)*N + j] + grid[(i+1)*N + j] +
                                    grid[i*N + (j-1)] + grid[i*N + (j+1)]);
    }
}

void laplace_cuda(std::vector<float>& grid, int N, int max_iter) {
    float *d_grid, *d_new_grid;
    hipError_t err;
    err = hipMalloc(&d_grid, N*N*sizeof(float));
    if (err != hipSuccess) { std::cerr << "CUDA: Failed to allocate d_grid\n"; return; }
    err = hipMalloc(&d_new_grid, N*N*sizeof(float));
    if (err != hipSuccess) { std::cerr << "CUDA: Failed to allocate d_new_grid\n"; hipFree(d_grid); return; }
    err = hipMemcpy(d_grid, grid.data(), N*N*sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) { std::cerr << "CUDA: Failed to copy to d_grid\n"; hipFree(d_grid); hipFree(d_new_grid); return; }

    dim3 block(16, 16);
    dim3 gridDim((N+block.x-1)/block.x, (N+block.y-1)/block.y);

    for (int it = 0; it < max_iter; ++it) {
        laplace_step<<<gridDim, block>>>(d_grid, d_new_grid, N);
        err = hipGetLastError();
        if (err != hipSuccess) { std::cerr << "CUDA: Kernel launch failed: " << hipGetErrorString(err) << "\n"; break; }
        std::swap(d_grid, d_new_grid);
    }
    err = hipMemcpy(grid.data(), d_grid, N*N*sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) { std::cerr << "CUDA: Failed to copy to host\n"; }
    hipFree(d_grid);
    hipFree(d_new_grid);
}
