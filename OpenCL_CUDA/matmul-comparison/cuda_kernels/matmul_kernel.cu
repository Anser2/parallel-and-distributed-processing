#include <hip/hip_runtime.h>
#include <iostream>

#define TILE_SIZE 16

__global__ void matmul_kernel(float* A, float* B, float* C, int N) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float value = 0.0f;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < N && t * TILE_SIZE + threadIdx.x < N) {
            tileA[threadIdx.y][threadIdx.x] = A[row * N + (t * TILE_SIZE + threadIdx.x)];
        } else {
            tileA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && t * TILE_SIZE + threadIdx.y < N) {
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            tileB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            value += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = value;
    }
}

void launch_matmul_kernel(float* A, float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threads(TILE_SIZE, TILE_SIZE);
    dim3 blocks((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);
    matmul_kernel<<<blocks, threads>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}