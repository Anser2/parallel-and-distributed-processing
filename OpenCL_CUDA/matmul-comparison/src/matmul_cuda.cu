#include "hip/hip_runtime.h"
// This file contains the CUDA version of matrix multiplication.
// It includes functions for setting up CUDA, launching kernels, and performing the multiplication on the GPU.

#include <iostream>
#include <hip/hip_runtime.h>
#include "matmul.hpp"

// CUDA kernel for matrix multiplication
__global__ void matmul_kernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

// Function to perform matrix multiplication using CUDA
void matmul_cuda(const float* h_A, const float* h_B, float* h_C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    // Allocate device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid sizes
    dim3 blockSize(16, 16); // 16x16 threads per block
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Launch the kernel
    matmul_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}